#include "hip/hip_runtime.h"
#include <wb.h>

//*** Macros ***//
#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define inBounds(x, y, z) \
  ((0 <= (x) && (x) < x_size) && \
   (0 <= (y) && (y) < y_size) && \
   (0 <= (z) && (z) < z_size))

//*** Program-wide constants ***//
#define KERNEL_SIZE   3
#define KERNEL_RADIUS 1

#define TILE_SIZE     KERNEL_SIZE
#define CACHE_SIZE    (KERNEL_SIZE + (KERNEL_RADIUS * 2))

//*** Device constant memory ***//
__constant__ float deviceKernel[KERNEL_SIZE * KERNEL_SIZE * KERNEL_SIZE];

//*** 3D convolution kernel ***//
__global__ void conv3d(float *input, float *output,
                       const int z_size, const int y_size, const int x_size) {

    // General Thread Info
    int bx = blockIdx.x * TILE_SIZE; int tx = threadIdx.x;
    int by = blockIdx.y * TILE_SIZE; int ty = threadIdx.y;
    int bz = blockIdx.z * TILE_SIZE; int tz = threadIdx.z;

    //*** Generate tileCache ***//
    __shared__ float tileCache[CACHE_SIZE][CACHE_SIZE][CACHE_SIZE];

    // map each thread to a position in the kernel
    int tid = tz * (KERNEL_SIZE * KERNEL_SIZE) + ty * (KERNEL_SIZE) + tx;
    if (tid < CACHE_SIZE * CACHE_SIZE) {

        // map each kernel position to location in tile cache
        int tileX =  tid % CACHE_SIZE;
        int tileY = (tid / CACHE_SIZE) % CACHE_SIZE;

        int inputX = bx + tileX - 1;
        int inputY = by + tileY - 1;
        int inputZPartial = bz - 1;
        int inputZ;

        // load part of the tile cache
        for (int i = 0; i < CACHE_SIZE; i += 1) {
            inputZ = inputZPartial + i;

            if (inBounds(inputX, inputY, inputZ)) {
                tileCache[tileX][tileY][i] = input[inputZ * (y_size * x_size) + inputY * (x_size) + inputX];
            } else {
                tileCache[tileX][tileY][i] = 0;
            }
        }
    }

    __syncthreads();

    //*** Perform block convolution ***//
    // Exit threads outside of matrix boundry
    int xPos = bx + tx;
    int yPos = by + ty;
    int zPos = bz + tz;

    if (inBounds(xPos, yPos, zPos)) {
        float outputValue = 0;
        for (int x = 0; x < KERNEL_SIZE; x += 1) {
            for (int y = 0; y < KERNEL_SIZE; y += 1) {
                for (int z = 0; z < KERNEL_SIZE; z += 1) {
                    outputValue +=
                            tileCache[tx + x][ty + y][tz + z] *
                            deviceKernel[z * (KERNEL_SIZE * KERNEL_SIZE) + y * (KERNEL_SIZE) + x];
                }
            }
        }
        output[zPos * (y_size * x_size) + yPos * (x_size) + xPos] = outputValue;
    }
}



int main(int argc, char *argv[]) {
    wbArg_t args;
    int z_size;
    int y_size;
    int x_size;
    int inputLength, kernelLength;
    float *hostInput;
    float *hostKernel;
    float *hostOutput;
    float *deviceInput;
    float *deviceOutput;
    args = wbArg_read(argc, argv);



    // Import data
    hostInput  = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostKernel = (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
    hostOutput = (float *)malloc(inputLength * sizeof(float));



    // First three elements are the input dimensions
    z_size = hostInput[0];
    y_size = hostInput[1];
    x_size = hostInput[2];
    wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
    assert(z_size * y_size * x_size == inputLength - 3);
    assert(kernelLength == 27);



    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");
    //*** Allocating GPU memory ***//
    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void**) &deviceInput,  z_size * y_size * x_size * sizeof(float));
    hipMalloc((void**) &deviceOutput, z_size * y_size * x_size * sizeof(float));
    wbTime_stop(GPU, "Doing GPU memory allocation");



    //*** Copying input memory to the GPU ***//
    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInput, hostInput + 3,  z_size * y_size * x_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(deviceKernel), hostKernel, kernelLength * sizeof(float), 0, hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");



    //*** Performing CUDA computation ***//
    wbTime_start(Compute, "Doing the computation on the GPU");
    dim3 dimGrid(ceil(x_size/double(TILE_SIZE)), ceil(y_size/double(TILE_SIZE)), ceil(z_size/double(TILE_SIZE)));
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, TILE_SIZE);
    conv3d<<<dimGrid, dimBlock>>>(
            deviceInput, deviceOutput,
            z_size, y_size, x_size
    );
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Doing the computation on the GPU");



    //*** Copying output memory to the CPU ***//
    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(hostOutput + 3, deviceOutput, z_size * y_size * x_size * sizeof(float), hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");
    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");



    // Set the output dimensions for correctness checking
    hostOutput[0] = z_size;
    hostOutput[1] = y_size;
    hostOutput[2] = x_size;



    //*** Check Solution ***//
    wbSolution(args, hostOutput, inputLength);



    //*** Freeing GPU Memory ***//
    hipFree(deviceInput);
    hipFree(deviceOutput);



    //*** Freeing CPU Memory ***//
    free(hostInput);
    free(hostOutput);



    //*** Exit ***//
    return 0;
}
