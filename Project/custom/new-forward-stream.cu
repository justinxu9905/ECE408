#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 8
#define STREAM_NUM 16

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int Width_grid = ceil(1.0 * Width_out / TILE_WIDTH);
    (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    (void)Width_out; // silence declared but never referenced warning. remove this line when you start working
    (void)Width_grid;

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

#define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
#define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
#define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here

    int b = blockIdx.x;
    int m = blockIdx.y;
    int h = blockIdx.z / Width_grid * TILE_WIDTH + threadIdx.y;
    int w = blockIdx.z % Width_grid * TILE_WIDTH + threadIdx.x;

    if (h < Height_out && w < Width_out) {
        float acc = 0.;
        for (int c = 0; c < Channel; c++) {
            for (int p = 0; p < K; p++) {
                for (int q = 0; q < K; q++) {
                    acc += in_4d(b, c, h+p, w+q) * mask_4d(m, c, p, q);
                }
            }
        }
        out_4d(b, m, h, w) = acc;
    }

#undef out_4d
#undef in_4d
#undef mask_4d
}


__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    hipStream_t streams[STREAM_NUM];

    hipMalloc((void **) device_input_ptr, Batch * Channel * Width * Height * sizeof(float));
    hipMalloc((void **) device_mask_ptr, Map_out * Channel * K * K * sizeof(float));
    hipMalloc((void **) device_output_ptr, Batch * Map_out * Width_out * Height_out * sizeof(float));

    for (int i = 0; i < STREAM_NUM; i++) {
        hipStreamCreate(&streams[i]);

        int input_stream_size = Batch * Channel * Width * Height * sizeof(float) / STREAM_NUM;
        int mask_stream_size = Map_out * Channel * K * K * sizeof(float) / STREAM_NUM;

        hipMemcpyAsync(&device_input_ptr[input_stream_size * i], &host_input[input_stream_size * i], input_stream_size, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(&device_mask_ptr[mask_stream_size * i], &host_mask[mask_stream_size * i], mask_stream_size, hipMemcpyHostToDevice, streams[i]);

        hipStreamDestroy(streams[i]);
    }

    // Useful snippet for error checking
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    const int Width_grid = ceil(1.0 * Width_out / TILE_WIDTH);
    const int Height_grid = ceil(1.0 * Height_out / TILE_WIDTH);
    const int Z = Width_grid * Height_grid;

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid(Batch, Map_out, Z);
    conv_forward_kernel<<<dimGrid, dimBlock>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    hipStream_t streams[STREAM_NUM];

    for (int i = 0; i < STREAM_NUM; i++) {
        hipStreamCreate(&streams[i]);

        int output_stream_size = Batch * Map_out * Width_out * Height_out * sizeof(float) / STREAM_NUM;

        hipMemcpyAsync(&host_output[output_stream_size * i], &device_output[output_stream_size * i], output_stream_size, hipMemcpyDeviceToHost, streams[i]);

        hipStreamDestroy(streams[i]);
    }

    // Free device memory

    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
